#include "hip/hip_runtime.h"
#include <cugemm.hpp>

template <class T>
__global__ void gemm_kernel(
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const T alpha,
		const T* const A_ptr, const unsigned lda,
		const T* const B_ptr, const unsigned ldb,
		const T beta,
		T* const C_ptr, const unsigned ldc
		) {
	const auto tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= m * n) {
		return;
	}
	const auto mi = tid % m;
	const auto ni = tid / m;

	T sum = 0;
	for (unsigned ki_global = 0; ki_global < k; ki_global += 8) {

		T local_sum = 0;
		unsigned ki;
		for (unsigned ki_local = 0; ki_local < 8 && (ki = ki_global + ki_local) < k; ki_local++) {
			const std::size_t A_offset = (op_A == HIPBLAS_OP_N ? (mi + ki * lda) : (ki + mi * lda));
			const std::size_t B_offset = (op_B == HIPBLAS_OP_N ? (ki + ni * ldb) : (ni + ki * ldb));
			local_sum += A_ptr[A_offset] * B_ptr[B_offset];
		}
		sum += local_sum;
	}

	if (beta == static_cast<T>(0)) {
		C_ptr[mi + ni * ldc] = alpha * sum;
	} else {
		C_ptr[mi + ni * ldc] = alpha * sum + beta * C_ptr[mi + ni * ldc];
	}
}

template <class T>
hipblasStatus_t mtk::cugemm::gemm(
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_A,
		hipblasOperation_t op_B,
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const T* const alpha,
		const T* const A_ptr, const unsigned lda,
		const T* const B_ptr, const unsigned ldb,
		const T* const beta,
		T* const C_ptr, const unsigned ldc
		) {
	const auto block_size = 256lu;
	const auto grid_size = (m * n + block_size - 1) / block_size;

	hipStream_t hip_stream = 0;
	if (cublas_handle != nullptr) {
		hipblasGetStream(cublas_handle, &hip_stream);
	}

	gemm_kernel<<<grid_size, block_size, 0, hip_stream>>>(
			op_A, op_B,
			m, n, k,
			*alpha,
			A_ptr, lda,
			B_ptr, ldb,
			*beta,
			C_ptr, ldc
			);

	return HIPBLAS_STATUS_SUCCESS;
}

#define GEMM_INSTANCE(T) \
template hipblasStatus_t mtk::cugemm::gemm<T>(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, const unsigned, const unsigned, const unsigned, const T* const, const T* const, const unsigned, const T* const, const unsigned, const T* const, T* const, const unsigned)
GEMM_INSTANCE(double);
GEMM_INSTANCE(float );
GEMM_INSTANCE(half  );
